#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCThrustAllocator.cuh"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

template <typename Dtype, typename Acctype>
struct smoothl1_functor
{
#if defined(__HIP_PLATFORM_HCC__)
  __host__ __device__
#endif
  smoothl1_functor() {}

  __host__ __device__ Acctype operator()(const Dtype &x, const Dtype &y) const
  {
    Acctype z = ScalarConvert<Dtype, Acctype>::to(THCNumerics<Dtype>::abs(x-y));
    return z < Acctype(1) ? 0.5f*z*z : z - 0.5f;
  }

#if defined(__HIP_PLATFORM_HCC__)
  __host__ __device__
  ~smoothl1_functor() {}
#endif
};

template <typename Dtype>
struct smoothl1_updateOutput_no_reduce_functor
{
  smoothl1_updateOutput_no_reduce_functor() {}

  __forceinline__ __host__ __device__ void operator()(
      const Dtype *x, 
      const Dtype *y,
      Dtype *out) const
  {
    Dtype oneHalf = ScalarConvert<float, Dtype>::to(0.5f);
    Dtype z = THCNumerics<Dtype>::abs(*x - *y);
    *out = z < ScalarConvert<int, Dtype>::to(1) ? oneHalf * z * z : z - oneHalf;
  }
};

template <typename Dtype>
struct smoothl1_updateGradInput_no_reduce_functor
{
  smoothl1_updateGradInput_no_reduce_functor() {}

  __host__ __device__ void operator()(
      const Dtype *x, 
      const Dtype *y,
      Dtype *gradInput) const
  {
    Dtype z = *x - *y;
    Dtype one = ScalarConvert<int, Dtype>::to(1);
    Dtype minusOne = ScalarConvert<int, Dtype>::to(-1);
    if (z < minusOne) {
      *gradInput = minusOne;
    } else if (z > one) {
      *gradInput = one;
    } else {
      *gradInput = z;
    }
  }
};

template <typename Dtype>
struct smoothl1_updateGradInput_functor
{
  const Dtype norm;
  const Dtype gradOutput;

#if defined(__HIP_PLATFORM_HCC__)
  __host__ __device__
  smoothl1_updateGradInput_functor() = default;

  __host__ __device__
  smoothl1_updateGradInput_functor(const smoothl1_updateGradInput_functor& f) = default;

  __host__ __device__
  ~smoothl1_updateGradInput_functor() {}

  __host__ __device__
  smoothl1_updateGradInput_functor(Dtype norm_, Dtype gradOutput_)
    : norm(norm_), gradOutput(gradOutput_)
  {}
#endif

  __host__ __device__ Dtype operator()(const Dtype &x, const Dtype &y) const
  {
    Dtype z = x - y;
    if (z < ScalarConvert<int, Dtype>::to(-1))
      return -norm * gradOutput;
    else if (z > ScalarConvert<int, Dtype>::to(1))
      return norm * gradOutput;
    else
      return norm * z * gradOutput;
  }
};

#include "generic/SmoothL1Criterion.cu"
#include "THCGenerateFloatTypes.h"
